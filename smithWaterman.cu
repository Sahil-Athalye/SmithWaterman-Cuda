#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <cctype>
#include <cstdio>
#include <algorithm>

// CUDA kernel to compute one anti-diagonal of the Smith-Waterman DP and direction matrices
__global__ void sw_kernel(const char *seq1, const char *seq2, int len1, int len2, 
                          int diag, int start_i, int end_i, 
                          int *score, unsigned char *dir, int matchScore, int mismatchScore, int gapScore) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int i = start_i + idx;
    if(i > end_i) return;
    int j = diag - i;
    // Compute scores for match/mismatch and gap options
    int up   = score[(i-1) * (len2+1) + j] + gapScore;
    int left = score[i * (len2+1) + (j-1)] + gapScore;
    int diagScore = score[(i-1) * (len2+1) + (j-1)] + ((seq1[i-1] == seq2[j-1]) ? matchScore : mismatchScore);
    // Choose the maximum, compare with 0 for local alignment
    int maxScore = 0;
    unsigned char direction = 0;
    if(diagScore > maxScore) {
        maxScore = diagScore;
        direction = 1; // 1 = diagonal
    }
    if(up > maxScore) {
        maxScore = up;
        direction = 2; // 2 = up (gap in seq2)
    }
    if(left > maxScore) {
        maxScore = left;
        direction = 3; // 3 = left (gap in seq1)
    }
    // Write back score and direction
    score[i * (len2+1) + j] = maxScore;
    dir[i * (len2+1) + j] = direction;
}

// Helper function to extract base name without directory or extension
std::string extractBaseName(const std::string& filepath) {
    // Find the last slash or backslash
    size_t lastSlash = filepath.find_last_of("/\\");
    size_t start = (lastSlash == std::string::npos) ? 0 : lastSlash + 1;
    
    // Find the last dot (extension)
    size_t lastDot = filepath.find_last_of('.');
    size_t end = (lastDot == std::string::npos || lastDot <= start) ? filepath.length() : lastDot;
    
    // Extract the filename without extension
    return filepath.substr(start, end - start);
}

int main(int argc, char **argv) {
    if(argc < 3) {
        std::cerr << "Usage: " << argv[0] << " <seq1.fasta> <seq2.fasta>\n";
        return 1;
    }
    std::string file1 = argv[1];
    std::string file2 = argv[2];

    // Read sequences from FASTA files
    std::ifstream fin1(file1);
    std::ifstream fin2(file2);
    if(!fin1.is_open() || !fin2.is_open()) {
        std::cerr << "Error: unable to open input FASTA file(s).\n";
        return 1;
    }
    std::string name1, name2;
    std::string seq1 = "", seq2 = "";
    std::string line;
    // Read first file
    if(std::getline(fin1, line)) {
        if(line.size() > 0 && line[0] == '>') {
            // Extract name (up to first whitespace or end of line after '>')
            size_t pos = 1;
            while(pos < line.size() && !isspace(static_cast<unsigned char>(line[pos]))) {
                name1.push_back(line[pos]);
                pos++;
            }
        }
        // Read sequence lines
        while(std::getline(fin1, line)) {
            if(line.size() > 0 && line[0] == '>') break; // stop if another header (should not happen for single sequence)
            for(char c : line) {
                if(!isspace(static_cast<unsigned char>(c))) {
                    seq1.push_back(c);
                }
            }
        }
    }
    // Read second file
    if(std::getline(fin2, line)) {
        if(line.size() > 0 && line[0] == '>') {
            size_t pos = 1;
            while(pos < line.size() && !isspace(static_cast<unsigned char>(line[pos]))) {
                name2.push_back(line[pos]);
                pos++;
            }
        }
        while(std::getline(fin2, line)) {
            if(line.size() > 0 && line[0] == '>') break;
            for(char c : line) {
                if(!isspace(static_cast<unsigned char>(c))) {
                    seq2.push_back(c);
                }
            }
        }
    }
    fin1.close();
    fin2.close();

    // If names are not provided in FASTA, use filenames instead
    if(name1.empty()) {
        name1 = extractBaseName(file1);
    }
    if(name2.empty()) {
        name2 = extractBaseName(file2);
    }

    // IMPORTANT: Strip any prefix like "BB11001_" from sequence names
    // This change ensures consistent sequence naming in output MSF files
    auto stripPrefix = [](const std::string& name) -> std::string {
        size_t pos = name.find('_');
        if(pos != std::string::npos && pos > 0) {
            return name.substr(pos + 1);
        }
        return name;
    };
    
    // Strip any prefixes from sequence names
    name1 = stripPrefix(name1);
    name2 = stripPrefix(name2);

    // Convert sequences to uppercase (for consistency in matching)
    for(char &c : seq1) {
        c = std::toupper(static_cast<unsigned char>(c));
    }
    for(char &c : seq2) {
        c = std::toupper(static_cast<unsigned char>(c));
    }

    int len1 = seq1.length();
    int len2 = seq2.length();
    if(len1 == 0 || len2 == 0) {
        std::cerr << "Error: one of the sequences is empty.\n";
        return 1;
    }

    // Allocate device memory
    char *d_seq1 = nullptr, *d_seq2 = nullptr;
    int *d_score = nullptr;
    unsigned char *d_dir = nullptr;
    size_t sizeScore = (size_t)(len1+1) * (len2+1) * sizeof(int);
    size_t sizeDir   = (size_t)(len1+1) * (len2+1) * sizeof(unsigned char);
    hipMalloc((void**)&d_seq1, len1 * sizeof(char));
    hipMalloc((void**)&d_seq2, len2 * sizeof(char));
    hipMalloc((void**)&d_score, sizeScore);
    hipMalloc((void**)&d_dir, sizeDir);
    // Copy sequences to device
    hipMemcpy(d_seq1, seq1.data(), len1 * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(d_seq2, seq2.data(), len2 * sizeof(char), hipMemcpyHostToDevice);
    // Initialize score and direction matrices to 0
    hipMemset(d_score, 0, sizeScore);
    hipMemset(d_dir,   0, sizeDir);

    // Scoring scheme (can be adjusted): match = +2, mismatch = -1, gap = -1
    int matchScore = 2;
    int mismatchScore = -1;
    int gapScore = -1;

    // Compute DP matrix anti-diagonal by anti-diagonal
    // Maximum possible diag index = len1 + len2 (when i=len1, j=len2)
    int maxDiag = len1 + len2;
    int threadsPerBlock = 256;
    for(int diag = 2; diag <= maxDiag; ++diag) {
        int start_i = (diag > len2+1) ? (diag - (len2+1) + 1) : 1;
        if(start_i < 1) start_i = 1;
        int end_i = (diag - 1 < len1) ? (diag - 1) : len1;
        if(end_i > len1) end_i = len1;
        if(start_i > len1 || start_i > end_i) continue; // no cells on this diag
        int totalCells = end_i - start_i + 1;
        int blocks = (totalCells + threadsPerBlock - 1) / threadsPerBlock;
        sw_kernel<<<blocks, threadsPerBlock>>>(d_seq1, d_seq2, len1, len2, diag, start_i, end_i, d_score, d_dir, matchScore, mismatchScore, gapScore);
        hipDeviceSynchronize();
    }

    // Copy score and direction matrices back to host
    std::vector<int> score((len1+1) * (len2+1));
    std::vector<unsigned char> dir((len1+1) * (len2+1));
    hipMemcpy(score.data(), d_score, sizeScore, hipMemcpyDeviceToHost);
    hipMemcpy(dir.data(), d_dir, sizeDir, hipMemcpyDeviceToHost);

    // Find the cell with maximum score for local alignment endpoint
    int maxScore = 0;
    int max_i = 0, max_j = 0;
    for(int i = 1; i <= len1; ++i) {
        for(int j = 1; j <= len2; ++j) {
            int val = score[i * (len2+1) + j];
            if(val > maxScore) {
                maxScore = val;
                max_i = i;
                max_j = j;
            }
        }
    }

    // Traceback from (max_i, max_j) until score becomes 0
    std::string align1 = "";
    std::string align2 = "";
    int ti = max_i;
    int tj = max_j;
    while(ti > 0 && tj > 0) {
        unsigned char d = dir[ti * (len2+1) + tj];
        if(d == 0) {
            break; // alignment stop
        }
        if(d == 1) { // diagonal
            align1.push_back(seq1[ti-1]);
            align2.push_back(seq2[tj-1]);
            ti -= 1;
            tj -= 1;
        } else if(d == 2) { // came from up (gap in seq2)
            align1.push_back(seq1[ti-1]);
            align2.push_back('-');  // use '-' for gap during traceback
            ti -= 1;
        } else if(d == 3) { // came from left (gap in seq1)
            align1.push_back('-');
            align2.push_back(seq2[tj-1]);
            tj -= 1;
        } else {
            // Should not happen for Smith-Waterman (d is 0-3)
            break;
        }
        if(score[ti * (len2+1) + tj] == 0) {
            // Stop when we hit a cell with 0 (beginning of local alignment)
            break;
        }
    }
    // Reverse the aligned strings as we collected them backward
    std::reverse(align1.begin(), align1.end());
    std::reverse(align2.begin(), align2.end());

    // Replace '-' gaps with '.' for MSF output format
    for(char &c : align1) {
        if(c == '-') c = '.';
    }
    for(char &c : align2) {
        if(c == '-') c = '.';
    }

    int alignLen = align1.size();
    // Compute checksum for each aligned sequence (GCG checksum)
    auto gcgChecksum = [](const std::string &s) {
        long check = 0;
        for(size_t i = 0; i < s.size(); ++i) {
            // use (i % 57) + 1 multiplier
            int c = std::toupper(static_cast<unsigned char>(s[i]));
            check += ((int)((i % 57) + 1) * c);
        }
        return (int)(check % 10000);
    };
    int check1 = gcgChecksum(align1);
    int check2 = gcgChecksum(align2);
    int globalCheck = ( (long)check1 + (long)check2 ) % 10000;

    // Determine sequence type for MSF (P for protein, N for nucleic acid) by scanning sequence letters
    char typeChar = 'P';
    std::string allAligned = align1 + align2;
    bool maybeDNA = true;
    for(char c : allAligned) {
        // skip gaps
        if(c == '.') continue;
        char u = std::toupper(static_cast<unsigned char>(c));
        if(u != 'A' && u != 'C' && u != 'G' && u != 'T' && u != 'U' && u != 'N') {
            maybeDNA = false;
            break;
        }
    }
    if(maybeDNA) typeChar = 'N';

    std::cout << "Alignment score: " << maxScore << "\n\n";

    // Output alignment in MSF (PileUp) format
    std::cout << "PileUp\n\n";
    std::printf("   MSF:   %d  Type: %c    Check:  %4d   ..\n\n", alignLen, typeChar, globalCheck);
    std::printf(" Name: %s oo  Len:   %d  Check:  %4d  Weight:  10.0\n", name1.c_str(), alignLen, check1);
    std::printf(" Name: %s oo  Len:   %d  Check:  %4d  Weight:  10.0\n\n", name2.c_str(), alignLen, check2);
    std::cout << "//\n\n";

    // Print aligned sequences in blocks of 50 columns
    int colsPerLine = 50;
    for(int start = 0; start < alignLen; start += colsPerLine) {
        int end = (start + colsPerLine < alignLen) ? (start + colsPerLine) : alignLen;
        // Sequence 1 line
        std::printf("%-12s", name1.c_str());  // name left padded to 12 characters
        // Print sequence with a space every 10 residues
        int count = 0;
        for(int k = start; k < end; ++k) {
            std::cout << align1[k];
            count++;
            if(count % 10 == 0 && k < end - 1) {
                std::cout << ' ';
            }
        }
        std::cout << "\n";
        // Sequence 2 line
        std::printf("%-12s", name2.c_str());
        count = 0;
        for(int k = start; k < end; ++k) {
            std::cout << align2[k];
            count++;
            if(count % 10 == 0 && k < end - 1) {
                std::cout << ' ';
            }
        }
        std::cout << "\n\n";
    }

    // Free device memory
    hipFree(d_seq1);
    hipFree(d_seq2);
    hipFree(d_score);
    hipFree(d_dir);
    return 0;
}